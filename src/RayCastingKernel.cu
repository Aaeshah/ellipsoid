#include "hip/hip_runtime.h"
#ifndef __RAYCASTINGKERNEL_CU__

#define __RAYCASTINGKERNEL_CU__

#include <GL/glew.h>

#include <hip/hip_vector_types.h>

#include "constants.h"

typedef unsigned int uint;

__global__ void render_elipsoid( GLubyte*screen , uint qw , uint qh , uint w , uint h , uint n )
{
	uint ix = qw * blockIdx.x + threadIdx.x;
	uint iy = qh * blockIdx.y;

	float A = .25;
	float B = .75;
	float C = .5;

	if( ix >= w ) return;

	float3 p  = make_float3( ((float)(blockIdx.x*qw)/(float)w - .5f)*2.0f ,
				 ((float)(blockIdx.y*qh)/(float)h - .5f)*2.0f ,
				 0.0f );

	float3 v  = make_float3( 0.0f , 0.0f , 1.0f );

	float AA = A*A;
	float BB = B*B;
	float CC = C*C;

	float a = -v.x*v.x*BB*CC - v.y*v.y*AA*CC - v.z*v.z*AA*BB;
	float b = -2.0f*( v.x*BB*CC*p.x + v.y*AA*CC*p.y + v.z*AA*BB*p.z );
	float c = AA*BB*CC - AA*BB*p.z*p.z - AA*CC*p.y*p.y - BB*CC*p.x*p.x;

	float d = b*b - 4*a*c;

	uchar3 color;

	if( d >= 0 ) {
		d = sqrt(d);
		float t = min( (-b+d)/(2.0f*a) , (-b-d)/(2.0f*a) );

		color = make_uchar3( 0 , -255.0f*t , 0 );
	} else	color = make_uchar3( 0 , 0 , 0 );


	uint idx;
	for( uint i = 0 ; i<qh ; i++ )
	{
		if( iy+i >= h ) return;

		idx = (ix + (iy+i)*w)*3;

		screen[idx  ] = color.x;
		screen[idx+1] = color.y;
		screen[idx+2] = color.z;
	}
}

#endif /* __RAYCASTINGKERNEL_CU__ */

