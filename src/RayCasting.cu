#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <GL/glew.h>
#include <GL/gl.h>

#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#include "RayCasting.h"
#include "RayCastingKernel.cu"

#include "logger.h"

#define A(row, col) a[(col << 2) + row]
#define B(row, col) b[(col << 2) + row]
#define P(row, col) product[(col << 2) + row]

void
matmul4(float       *res,
        const float *a,
        const float *b)
{
	float product[16];
	int i;

	for (i = 0; i < 4; i++)
	{
		const float ai0 = A(i, 0), ai1 = A(i, 1), ai2 = A(i, 2), ai3 = A(i, 3);

		P(i, 0) = ai0 * B(0, 0) + ai1 * B(1, 0) + ai2 * B(2, 0) + ai3 * B(3, 0);
		P(i, 1) = ai0 * B(0, 1) + ai1 * B(1, 1) + ai2 * B(2, 1) + ai3 * B(3, 1);
		P(i, 2) = ai0 * B(0, 2) + ai1 * B(1, 2) + ai2 * B(2, 2) + ai3 * B(3, 2);
		P(i, 3) = ai0 * B(0, 3) + ai1 * B(1, 3) + ai2 * B(2, 3) + ai3 * B(3, 3);
	}
	
	memcpy( res , product , sizeof(float)*16 );
}

RayCasting::RayCasting( float a , float b , float c , float _m )
	: width(0) , height(0) , step(0) , d_m(NULL)
{
	e.a = a;
	e.b = b;
	e.c = c;
	e.m =_m;

	float i[16] = { 1, 0, 0, 0,
			0, 1, 0, 0,
			0, 0, 1, 0,
			0, 0, 0, 1 };

	memcpy( m , i , sizeof(float)*16 );
}

RayCasting::~RayCasting()
{
	if( d_m ) {
		hipFree( d_m );
		CUT_CHECK_ERROR("RayCasting::~RayCasting::hipFree");
	}
}

void RayCasting::translate( float x , float y , float z )
{
	float t[16] = { 1, 0, 0, x,
			0, 1, 0, y,
			0, 0, 1, z,
			0, 0, 0, 1 };

	matmul4( m  , t , m);
}

void RayCasting::scale( float x , float y , float z )
{
	float s[16] = { x, 0, 0, 0,
			0, y, 0, 0,
			0, 0, z, 0,
			0, 0, 0, 1 };

	matmul4( m , s , m );
}

void RayCasting::rotate( float a , float x , float y , float z )
{
	float c = cos( a );
	float s = sin( a );
	float xx = x*x;
	float yy = y*y;
	float zz = z*z;

	float r[16] = {  xx+(1-xx)*c  , x*y*(1-c)-z*s , x*z*(1-c)+y*s , 0 ,
			x*y*(1-c)+z*s ,  yy+(1-yy)*c  , y*z*(1-c)-x*s , 0 ,
			x*z*(1-c)-y*s , y*z*(1-c)+x*s ,  zz+(1-zz)*c  , 0 ,
			      0       ,       0       ,       0       , 1 };

	matmul4( m , r , m );
}

void RayCasting::resize( int w , int h )
{
	if( !d_m ) { 
		hipMalloc( (void**)&d_m , sizeof(float)*16 );
		CUT_CHECK_ERROR("RayCasting::RayCasting::hipMalloc");
	}

	width = w; height = h;

	GLubyte*d_ub;
	cudaGLMapBufferObject( (void**)&d_ub , pbo.pbo );
	CUT_CHECK_ERROR("RayCasting::init::cudaGLMapBufferObject");

	hipMemset( d_ub , 0 , sizeof(GLubyte)*w*h*3 );
	CUT_CHECK_ERROR("RayCasting::init::hipMemset");

	cudaGLUnmapBufferObject( pbo.pbo );
	CUT_CHECK_ERROR("RayCasting::init::cudaGLUnmapBufferObject");
}

bool RayCasting::render_frame( bool next )
{
	unsigned int quads = pow(2,step);

	if( next && (quads < width || quads < height) )
		++step;

	dim3 threads;
	while( (threads = std::ceil( (float)width / (float)(quads=pow(2,step)))).x >= 512 ) ++step;

	unsigned int qw = quads , qh = quads;
	while( qw > width ) qw >>= 1;
	while( qh > height) qh >>= 1;
	qw <<= 1 ; qh <<= 1;
	dim3 blocks  = dim3( quads , qh );

	GLubyte*d_ub;

	cudaGLMapBufferObject( (void**)&d_ub , pbo.pbo );
	CUT_CHECK_ERROR("RayCasting::init::cudaGLMapBufferObject");

	log_printf(DBG,"width %d\theight %d\n",width,height);
	log_printf(DBG,"thr: %d\tblk: %d %d\n",threads.x,blocks.x,blocks.y);

/*        for( int i=0 ; i<16 ; i++ ) printf("%f%c",m[i],i%4-3?' ':'\n');*/
/*        printf("\n");*/

	hipMemcpy( (void**)d_m , (void**)m , sizeof(float)*16 , hipMemcpyHostToDevice );
	CUT_CHECK_ERROR("RayCasting::render_frame::hipMemcpy");

	hipMemset( d_ub , 0 , sizeof(GLubyte)*width*height*3 );
	CUT_CHECK_ERROR("RayCasting::render_frame::hipMemset");

	render_elipsoid<<< blocks , threads >>>( d_ub , std::ceil( (float)width / (float)quads  ) , std::ceil( (float)height / (float)quads  ), width , height , e , d_m );
	CUT_CHECK_ERROR("RayCasting::render_frame::render_elipsoid");

	cudaGLUnmapBufferObject( pbo.pbo );
	CUT_CHECK_ERROR("RayCasting::render_frame::cudaGLUnmapBufferObject");

	if( quads < width || quads < height )
		return false;

	return true;
}

